#include "hip/hip_runtime.h"
#pragma once

#define MAX(a, b) (((a) >= (b)) ? (a) : (b))
#define MIN(a, b) (((a) <= (b)) ? (a) : (b))

#define ROUND_UP(a, b) (~((b) - 1) & ((a) + (b) - 1))
#define ROUND_DOWN(a, b) (~(b - 1) & a)

#define DIV_UP(a, b) (((a) + (b) - 1) / (b))

#define WARP_SIZE 32
#define LOG_WARP_SIZE 5


#ifndef NUM_WARPS
#define NUM_WARPS (NUM_THREADS / WARP_SIZE)
#else
#define NUM_THREADS (NUM_WARPS * WARP_SIZE)
#endif

#ifndef NO_VIDEO_INSTRUCTIONS
#define USE_VIDEO_INSTRUCTIONS
#endif

#include <hip/device_functions.h>
#include <vector_functions.h>
#include <sm_11_atomic_functions.h>		// atomicAdd()

#define DEVICE extern "C" __device__ __forceinline__
#define DEVICE2 __device__ __forceinline__

typedef unsigned int uint;
typedef unsigned short uint16;


// retrieve numBits bits from x starting at bit
DEVICE uint bfe(uint x, uint bit, uint numBits) {
	uint ret;
#if __CUDA_ARCH__ >= 200
	asm("bfe.u32 %0, %1, %2, %3;" : "=r"(ret) : "r"(x), "r"(bit), "r"(numBits));
#else
	uint mask = (1 << numBits) - 1;
	ret = (x >> bit) & mask;
#endif
	return ret;
}


// insert the first numBits of y into x starting at bit
DEVICE uint bfi(uint x, uint y, uint bit, uint numBits) {
	uint ret;
#if __CUDA_ARCH__ >= 200
	asm("bfi.b32 %0, %1, %2, %3, %4;" : 
		"=r"(ret) : "r"(y), "r"(x), "r"(bit), "r"(numBits));
#else
	uint mask = (1 << numBits) - 1;
	ret = (x & ~(mask << bit)) | ((y & mask) << bit);
#endif
	return ret;
}

// Same syntax as __byte_perm, but without nvcc's __byte_perm bug that masks all
// non-immediate index arguments by 0x7777.
DEVICE uint prmt(uint a, uint b, uint index) {
	uint ret;
#if __CUDA_ARCH__ >= 200
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(ret) : "r"(a), "r"(b), "r"(index));
#else
	ret = __byte_perm(a, b, index);
#endif
	return ret;
}

DEVICE uint shl_add(uint a, uint b, uint c) {
#if defined(USE_VIDEO_INSTRUCTIONS) && __CUDA_ARCH__ >= 200
	uint ret;
	asm("vshl.u32.u32.u32.clamp.add %0, %1, %2, %3;" :
		"=r"(ret) : "r"(a), "r"(b), "r"(c));
	return ret;
#else
	return (a<< b) + c;
#endif
}

// (a<< b) + c, where b is a constant. We hope to use the ISCADD instruction 
// rather than the vshl.add instruction.
DEVICE uint shl_add_c(uint a, uint b, uint c) {
	return (a<< b) + c;
}

DEVICE uint shr_add(uint a, uint b, uint c) {
#if defined(USE_VIDEO_INSTRUCTIONS) && __CUDA_ARCH__ >= 200
	uint ret;
	asm("vshr.u32.u32.u32.clamp.add %0, %1, %2, %3;" : 
		"=r"(ret) : "r"(a), "r"(b), "r"(c));
	return ret;
#else
	return (a>> b) + c;
#endif
}

DEVICE uint mul_add(uint a, uint b, uint c) {
#if defined(USE_VIDEO_INSTRUCTIONS) && __CUDA_ARCH__ >= 200
	uint ret;
	asm("vmad.u32.u32.u32 %0, %1, %2, %3;" : 
		"=r"(ret) : "r"(a), "r"(b), "r"(c));
	return ret;
#else
	return (a * b) + c;
#endif
}

DEVICE uint imad(uint a, uint b, uint c) {
#if defined(USE_VIDEO_INSTRUCTIONS) && __CUDA_ARCH__ >= 200
	uint ret;
	asm("mad.lo.u32 %0, %1, %2, %3;" : "=r"(ret) : "r"(a), "r"(b), "r"(c));
	return ret;
#else
	return a * b + c;
#endif
}


DEVICE uint GetByte(uint a, uint i) {
	return prmt(a, 0, 0x4440 + i);
}
DEVICE uint ExpandUint8Low(uint a) {
	return prmt(a, 0, 0x4140);
}
DEVICE uint ExpandUint8High(uint a) {
	return prmt(a, 0, 0x4342);
}
DEVICE uint2 Expand8Uint4To8Uint8(uint a) {
	// b.x = (0xf & a) | // 0 -> 0
	// ((0xf0 & a) << 4) | // 4 -> 8
	// ((0xf00 & a) << 8) | // 8 -> 16
	// ((0xf000 & a) << 12); // 12 -> 24
	// b.y = ((0xf0000 & a) >> 16) | // 16 -> 0
	// ((0xf00000 & a) >> 12) | // 20 -> 8
	// ((0xf000000 & a) >> 8) | // 24 -> 16
	// ((0xf0000000 & a) >> 4); // 28 -> 24
	uint2 b;
	uint a2 = a>> 4;
	b.x = 0x0f0f0f0f & prmt(a, a2, 0x5140);
	b.y = 0x0f0f0f0f & prmt(a, a2, 0x7362);
	return b;
}

DEVICE uint StridedThreadOrder(uint index) {
	return index + (index / WARP_SIZE);
}


DEVICE uint LoadKey(const uint* keys_global, uint index, uint numElements, 
	bool checkRange) {
	uint key = 0xffffffff;
	if(checkRange)
		if(index < numElements) key = keys_global[index];
	else
		key = keys_global[index];
	return key;
}

// volatile qualifier appropriate for shared memory.
DEVICE2 uint LoadShifted(const volatile uint* shared, uint shiftedIndex) {
	return *((volatile uint*)(((volatile char*)shared) + shiftedIndex));
}
DEVICE2 void StoreShifted(volatile uint* shared, uint shiftedIndex, uint val) {
	*((volatile uint*)(((volatile char*)shared) + shiftedIndex)) = val;
}


// Put a float into radix order.
DEVICE float UintToFloat(uint u) {
	int adjusted = (int)u;
	
	// Negative now has high bit set, positive has high bit clear.
	int flipped = adjusted - 0x80000000;
	
	// Fill the register with set bits if negative.	
	int bits = flipped>> 31;

	int x = flipped ^ (0x7fffffff & bits);

	float f = __int_as_float(x);
	return f;
}

// Put a radix order into back into a float.
DEVICE uint FloatToUint(float f) {
	int x = __float_as_int(f);
	int bits = x>> 31;

	int flipped = x ^ (0x7fffffff & bits);

	int adjusted = 0x80000000 + flipped;

	uint u = (uint)adjusted;
	return u;
}
