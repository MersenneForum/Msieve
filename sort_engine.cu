#include <stdio.h>
#include <b40c/util/error_utils.cuh>
#include <b40c/util/multi_buffer.cuh>
#include <b40c/radix_sort/enactor.cuh>

#include "sort_engine.h"

typedef unsigned int uint32;

#if defined(_WIN32) || defined (_WIN64)
	#define SORT_ENGINE_DECL __declspec(dllexport)
	typedef unsigned __int64 uint64;
#else
	#define SORT_ENGINE_DECL __attribute__((visibility("default")))
	typedef unsigned long long uint64;
#endif

using namespace b40c;

typedef struct
{
	radix_sort::Enactor enactor;
} sort_engine;

extern "C"
{

SORT_ENGINE_DECL void * 
sort_engine_init(void)
{
	return new sort_engine;
}

SORT_ENGINE_DECL void 
sort_engine_free(void * e)
{
	delete (sort_engine *)e;
}

SORT_ENGINE_DECL void 
sort_engine_run(void * e, sort_data_t * data)
{
	sort_engine *engine = (sort_engine *)e;
	bool need_swap;

	// arrays are assumed packed together; check
	// they would all start on a power-of-two boundary

	if (data->num_arrays > 1 && data->num_elements % 16) {
		printf("sort_engine: invalid array size\n");
		exit(-1);
	}

	if (data->key_bits == 64) {
		for (size_t i = 0; i < data->num_arrays; i++) {

			hipError_t status;
			util::DoubleBuffer<uint64, uint32> ptrs;

			ptrs.d_keys[0] = (uint64 *)data->keys_in +
						i * data->num_elements;
			ptrs.d_keys[1] = (uint64 *)data->keys_in_scratch +
						i * data->num_elements;
			ptrs.d_values[0] = (uint32 *)data->data_in +
						i * data->num_elements;
			ptrs.d_values[1] = (uint32 *)data->data_in_scratch +
						i * data->num_elements;

			status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		64, 0>(ptrs, data->num_elements);
			need_swap = (ptrs.selector > 0);
			if (status != hipSuccess) {
				util::B40CPerror(status, "sort engine: ", 
						__FILE__, __LINE__);
				exit(-1);
			}
		}
	}
	else {
		for (size_t i = 0; i < data->num_arrays; i++) {

			hipError_t status;
			util::DoubleBuffer<uint32, uint32> ptrs;

			ptrs.d_keys[0] = (uint32 *)data->keys_in +
						i * data->num_elements;
			ptrs.d_keys[1] = (uint32 *)data->keys_in_scratch +
						i * data->num_elements;
			ptrs.d_values[0] = (uint32 *)data->data_in +
						i * data->num_elements;
			ptrs.d_values[1] = (uint32 *)data->data_in_scratch +
						i * data->num_elements;

			status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		32, 0>(ptrs, data->num_elements);
			need_swap = (ptrs.selector > 0);
			if (status != hipSuccess) {
				util::B40CPerror(status, "sort engine: ", 
						__FILE__, __LINE__);
				exit(-1);
			}
		}
	}

	if (need_swap == true) {
		std::swap(data->keys_in, data->keys_in_scratch);
		std::swap(data->data_in, data->data_in_scratch);
	}
}

} // extern "C"
